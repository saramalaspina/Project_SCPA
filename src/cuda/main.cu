#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "../../lib/utils.h"

int main(int argc, char *argv[]) {

    if (argc < 2) {
        fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
        exit(1);
    }    

    // Read the matrix from a Matrix Market file
    MatrixElement *mat = read_matrix(argv[1]);
    if (!mat) exit(1);

    int rows = mat->M;
    int cols = mat->N;
    int nz = mat->nz;
    char *matrix_name;

    // Extract matrix name from the file path
    matrix_name = strrchr(argv[1], '/');
    if (matrix_name != NULL) {
        matrix_name++; 
    } else {
        matrix_name = argv[1];
    }

    char *dot = strrchr(matrix_name, '.');
    if (dot != NULL) {
        *dot = '\0';
    }
    
    // Generate input vector x
    double *x = generate_vector(matrix_name, cols);

    // Performance measurement variables
    clock_t start_time, end_time;
    int i;

    // Allocate memory for storing multiple execution times
    double *times = (double *)malloc(REPETITIONS * sizeof(double));
    if (times == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }

    // Allocate memory for average execution time of each version
    double *time_serial = (double *) malloc(sizeof(double));
    if(time_serial == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }

    double *time_csr = (double *) malloc(sizeof(double));
    if(time_csr == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }

    double *time_hll = (double *) malloc(sizeof(double));
    if(time_hll == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }

    // Sort the matrix in COO format
    qsort(mat->matrix, nz, sizeof(COOElement), compare_coo);
    
    // Convert matrix from COO to CSR format
    CSRMatrix *csr = convert_coo_to_csr(mat->matrix, nz, rows);

    // Allocate output vector for the serial result
    double *y_serial = (double *)calloc(rows, sizeof(double)); 
    if (!y_serial) {
        fprintf(stderr, "Memory allocation error\n");
        exit(1);
    }

    // Run the serial version and measure execution time
    for (i = 0; i < REPETITIONS; i++) {
        start_time = clock();
        prod_serial(rows, csr, x, y_serial);
        end_time = clock();
        times[i] = ((double)(end_time - start_time) / CLOCKS_PER_SEC) * 1000;
    }

    calculate_performance_cuda(times, mat, matrix_name, "serial", time_serial);
    
    // Reset the times array
    memset(times, 0, REPETITIONS * sizeof(double));

    // Allocate output vector for CUDA CSR result
    double *y_csr = (double *)calloc(rows, sizeof(double)); 
    if (!y_csr) {
        fprintf(stderr, "Memory allocation error\n");
        exit(1);
    }

    // Allocate memory for CUDA elapsed time
    float *elapsed_time_csr = (float *) malloc(sizeof(float));
    if(elapsed_time_csr == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }
    
    // Run CUDA CSR version and measure execution time
    for (i = 0; i < REPETITIONS; i++) {
        prod_cuda_csr(rows, cols, csr, x, y_csr, elapsed_time_csr);
        times[i] = *elapsed_time_csr;
    }

    // Verify correctness of CSR CUDA result
    if(check_results(y_serial, y_csr, rows) == 0){
        printf("Serial result is different from parallel result with csr\n");
    } else {
        printf("CSR results checked\n");
    }

    // Calculate average time for CUDA CSR
    calculate_performance_cuda(times, mat, matrix_name, "CSR", time_csr);

    // Reset times array
    memset(times, 0, REPETITIONS * sizeof(double));

    // Free CSR-related resources
    free(y_csr);
    free(elapsed_time_csr);
    free_csr_matrix(csr);

    // Convert matrix from COO to HLL format
    HLLMatrix *hll = convert_coo_to_hll(mat, HACKSIZE);

    // Allocate output vector for CUDA HLL result
    double *y_hll = (double *)calloc(rows, sizeof(double)); 
    if (!y_hll) {
        fprintf(stderr, "Memory allocation error\n");
        exit(1);
    }

    // Allocate memory for CUDA HLL elapsed time
    float *elapsed_time_hll = (float *) malloc(sizeof(float));
    if(elapsed_time_hll == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }
    
    // Run CUDA HLL version and measure execution time
    for (i = 0; i < REPETITIONS; i++) {
        prod_cuda_hll(hll, x, y_hll, rows, elapsed_time_hll);
        times[i] = *elapsed_time_hll;
    }

    // Verify correctness of HLL CUDA result
    if(check_results(y_serial, y_hll, rows) == 0){
        printf("Serial result is different from parallel result with hll\n");
    } else {
        printf("HLL results checked\n");
    }

    calculate_performance_cuda(times, mat, matrix_name, "HLL", time_hll);

    // Compute and save speedup results
    calculate_speedup(matrix_name, *time_serial, *time_csr, *time_hll, "results/cuda/speedup.csv", 0, nz);

    // Free all allocated memory
    free(times);
    free(time_serial);
    free(time_csr);
    free(time_hll);
    free(elapsed_time_hll);
    free(y_serial);
    free(y_hll);
    free_hll_matrix(hll);
    free(x);
    free(mat->matrix);
    free(mat);

    return 0;
}
