#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "../../lib/utils.h"

int main(int argc, char *argv[]) {

    if (argc < 2) {
        fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
        exit(1);
    }    

    MatrixElement *mat = read_matrix(argv[1]);
    if (!mat) exit(1);

    int rows = mat->M;
    int cols = mat->N;
    int nz = mat->nz;
    char *matrix_name;

    matrix_name = strrchr(argv[1], '/');
    if (matrix_name != NULL) {
        matrix_name++; 
    } else {
        matrix_name = argv[1];
    }

    char *dot = strrchr(matrix_name, '.');
    if (dot != NULL) {
        *dot = '\0';
    }
    
    //Creazione vettore x
    double *x = generate_vector(matrix_name, cols);

    //Variabili per misura delle prestazioni
    clock_t start_time, end_time;
    int i;

    double *times = (double *)malloc(REPETITIONS * sizeof(double));
    if (times == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }

    double *time_serial = (double *) malloc(sizeof(double));
    if(time_serial == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }

    double *time_csr = (double *) malloc(sizeof(double));
    if(time_csr == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }

    double *time_hll = (double *) malloc(sizeof(double));
    if(time_hll == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }

    qsort(mat->matrix, nz, sizeof(COOElement), compare_coo);
    
    //Creazione struct formato CSR
    CSRMatrix *csr = convert_coo_to_csr(mat->matrix, nz, rows);

    //Allocazione risultato seriale
    double *y_serial = (double *)calloc(rows, sizeof(double)); 
    if (!y_serial) {
        fprintf(stderr, "Memory allocation error\n");
        exit(1);
    }

    //Calcolo seriale e misura dei tempi
    for (i = 0; i < REPETITIONS; i++) {
        start_time = clock();
        prod_serial(rows, csr, x, y_serial);
        end_time = clock();
        times[i] = ((double)(end_time - start_time) / CLOCKS_PER_SEC) * 1000;
    }

    // printResult(y_serial, rows);

    calculate_performance_cuda(times, mat, matrix_name, "serial", time_serial);
    
    memset(times, 0, REPETITIONS * sizeof(double));

    //Allocazione risultato CUDA CSR
    double *y_csr = (double *)calloc(rows, sizeof(double)); 
    if (!y_csr) {
        fprintf(stderr, "Memory allocation error\n");
        exit(1);
    }

    float *elapsed_time_csr = (float *) malloc(sizeof(float));
    if(elapsed_time_csr == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }
    
    for (i = 0; i < REPETITIONS; i++) {
        prod_cuda_csr(rows, cols, csr, x, y_csr, elapsed_time_csr);
        times[i] = *elapsed_time_csr;
    }

    if(check_results(y_serial, y_csr, rows) == 0){
        printf("Serial result is different from parallel result with csr\n");
    } else {
        printf("CSR results checked\n");
    }

    calculate_performance_cuda(times, mat, matrix_name, "CSR", time_csr);

    memset(times, 0, REPETITIONS * sizeof(double));

    free(y_csr);
    free(elapsed_time_csr);
    free_csr_matrix(csr);

    // Creazione struct formato HLL
    HLLMatrix *hll = convert_coo_to_hll(mat, HACKSIZE) ;

    //Allocazione risultato Openmp HLL 
    double *y_hll = (double *)calloc(rows, sizeof(double)); 
    if (!y_hll) {
        fprintf(stderr, "Memory allocation error\n");
        exit(1);
    }

    float *elapsed_time_hll = (float *) malloc(sizeof(float));
    if(elapsed_time_hll == NULL) {
        fprintf(stderr, "Memory allocation error\n");
        exit(EXIT_FAILURE);
    }
    
    for (i = 0; i < REPETITIONS; i++) {
        prod_cuda_hll(hll, x, y_hll, rows, elapsed_time_hll);
        times[i] = *elapsed_time_hll;
    }

    if(check_results(y_serial, y_hll, rows) == 0){
        printf("Serial result is different from parallel result with hll\n");
    } else {
        printf("HLL results checked\n");
    }

    calculate_performance_cuda(times, mat, matrix_name, "HLL", time_hll);

    calculate_speedup(matrix_name, *time_serial, *time_csr, *time_hll, "results/cuda/speedup.csv", 0);

    free(times);
    free(time_serial);
    free(time_csr);
    free(time_hll);
    free(elapsed_time_hll);
    free(y_serial);
    free(y_hll);
    free_hll_matrix(hll);
    free(x);
    free(mat->matrix);
    free(mat);

    return 0;
}
