#include "hip/hip_runtime.h"
#include "../../lib/utils.cu"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>


__global__ void spmv_csr_kernel(int M, int *IRP, int *JA, double *AS, double *x, double *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M) {
        double sum = 0.0;
        for (int j = IRP[row]; j < IRP[row + 1]; j++) {
            sum += AS[j] * x[JA[j]];
        }
        y[row] = sum;
    }
}

double *spmv_csr_cuda(int M, int *IRP, int *JA, double *AS, double *x) {
    double *y = (double *)malloc(M * sizeof(double));
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;
    
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, IRP[M] * sizeof(int));
    hipMalloc(&d_AS, IRP[M] * sizeof(double));
    hipMalloc(&d_x, M * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));
    
    hipMemcpy(d_IRP, IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_JA, JA, IRP[M] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_AS, AS, IRP[M] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, M * sizeof(double), hipMemcpyHostToDevice);
    
    int blocks = (M + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    spmv_csr_kernel<<<blocks, THREADS_PER_BLOCK>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);
    
    hipMemcpy(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);
    
    return y;
}

__global__ void spmv_hll_kernel(int total_rows, int max_nz, double *AS, int *JA, double *x, double *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < total_rows) {
        double sum = 0.0;
        for (int j = 0; j < max_nz; j++) {
            sum += AS[row * max_nz + j] * x[JA[row * max_nz + j]];
        }
        y[row] = sum;
    }
}

double *spmv_hll_cuda(int total_rows, int max_nz, double *AS, int *JA, double *x) {
    double *y = (double *)malloc(total_rows * sizeof(double));
    double *d_AS, *d_x, *d_y;
    int *d_JA;
    
    hipMalloc(&d_AS, total_rows * max_nz * sizeof(double));
    hipMalloc(&d_JA, total_rows * max_nz * sizeof(int));
    hipMalloc(&d_x, total_rows * sizeof(double));
    hipMalloc(&d_y, total_rows * sizeof(double));
    
    hipMemcpy(d_AS, AS, total_rows * max_nz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_JA, JA, total_rows * max_nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, total_rows * sizeof(double), hipMemcpyHostToDevice);
    
    int blocks = (total_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    spmv_hll_kernel<<<blocks, THREADS_PER_BLOCK>>>(total_rows, max_nz, d_AS, d_JA, d_x, d_y);
    
    hipMemcpy(y, d_y, total_rows * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_AS);
    hipFree(d_JA);
    hipFree(d_x);
    hipFree(d_y);
    
    return y;
}
