#include "hip/hip_runtime.h"
#include "../../lib/utils.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

//CSR

__global__ void spmv_csr_kernel(int M, int *IRP, int *JA, double *AS, double *x, double *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M) {
        double sum = 0.0;
        for (int j = IRP[row]; j < IRP[row + 1]; j++) {
            sum += AS[j] * x[JA[j]];
        }
        y[row] = sum;
    }
}

double *prodCudaCSR(int M, int N, int *IRP, int *JA, double *AS, double *x) {
    double *y = (double *)malloc(M * sizeof(double));
    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;
    
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, IRP[M] * sizeof(int));
    hipMalloc(&d_AS, IRP[M] * sizeof(double));
    hipMalloc(&d_x, N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));
    
    hipMemcpy(d_IRP, IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_JA, JA, IRP[M] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_AS, AS, IRP[M] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);

    int blocks = (M + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    spmv_csr_kernel<<<blocks, THREADS_PER_BLOCK>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize(); 
    
    hipMemcpy(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);
    
    return y;
}

//HLL

__global__ void spmv_hll_kernel(int rows, int max_nz, const int *JA_t, const double *AS_t, const double *x, double *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        double sum = 0.0;
        // Per ogni "colonna" (cioè ogni posizione nella riga ELLPACK)
        for (int j = 0; j < max_nz; j++) {
            // Poiché i dati sono trasposti, l’accesso è:
            // elemento (row,j) in formato ELLPACK --> JA_t[j*rows + row] e AS_t[j*rows + row]
            int col = JA_t[j * rows + row];
            double val = AS_t[j * rows + row];
            sum += val * x[col];
        }
        y[row] = sum;
    }
}

double *prodCudaHLL(const HLLMatrix *hll, int total_rows, int total_cols, const double *x) {
    // Allocazione del vettore risultato sul host
    double *y = (double*)malloc(total_rows * sizeof(double));

    // Allocazione e copia del vettore x sul device
    double *d_x, *d_y;
    hipMalloc(&d_x, total_cols * sizeof(double));
    hipMalloc(&d_y, total_rows * sizeof(double));
    hipMemcpy(d_x, x, total_cols * sizeof(double), hipMemcpyHostToDevice);

    int row_offset = 0;  // per posizionare i risultati parziali all’interno di y

    // Processa ciascun blocco HLL (ognuno contiene un blocco in formato ELLPACK)
    for (int b = 0; b < hll->num_blocks; b++) {
        // Puntatore al blocco corrente
        ELLBlock *block = &(hll->blocks[b]);
        int rows = block->rows;
        int max_nz = block->max_nz;

        // Dimensione dei dati trasposti per il blocco
        size_t size_int = rows * max_nz * sizeof(int);
        size_t size_double = rows * max_nz * sizeof(double);

        // Alloca memoria sul device per JA_t e AS_t del blocco
        int *d_JA_t;
        double *d_AS_t;
        hipMalloc(&d_JA_t, size_int);
        hipMalloc(&d_AS_t, size_double);

        // Copia dei dati del blocco sul device
        hipMemcpy(d_JA_t, block->JA_t, size_int, hipMemcpyHostToDevice);
        hipMemcpy(d_AS_t, block->AS_t, size_double, hipMemcpyHostToDevice);

        // Calcola la configurazione di esecuzione per il kernel
        int numBlocks = (rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        // Il kernel scrive in d_y a partire da d_y + row_offset
        spmv_hll_kernel<<<numBlocks, THREADS_PER_BLOCK>>>(rows, max_nz, d_JA_t, d_AS_t, d_x, d_y + row_offset);
        hipDeviceSynchronize();

        // Libera la memoria allocata per il blocco
        hipFree(d_JA_t);
        hipFree(d_AS_t);

        row_offset += rows;
    }

    // Copia del vettore risultato dal device al host
    hipMemcpy(y, d_y, total_rows * sizeof(double), hipMemcpyDeviceToHost);

    // Libera la memoria sul device
    hipFree(d_x);
    hipFree(d_y);

    return y;
}




