#include "hip/hip_runtime.h"
#include "../../lib/utils.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define MAX_NZ_PER_ROW 256

//CSR

__global__ void spmv_csr_kernel(int M, int *IRP, int *JA, double *AS, double *x, double *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M) {
        double sum = 0.0;
        for (int j = IRP[row]; j < IRP[row + 1]; j++) {
            sum += AS[j] * x[JA[j]];
        }
        y[row] = sum;
    }
}

//CSR con Warp 

__global__ void spmv_csr_warp_kernel(int M, int *IRP, int *JA, double *AS, double *x, double *y) {
    int row = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int lane = threadIdx.x % WARP_SIZE;

    if (row < M) {
        double sum = 0.0;
        int row_start = IRP[row];
        int row_end = IRP[row + 1];
        
        for (int j = row_start + lane; j < row_end; j += WARP_SIZE) {
            sum += AS[j] * x[JA[j]];
        }
        
        for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }
        
        if (lane == 0) y[row] = sum;
    }     
}

void prod_cuda_csr(int M, int N, CSRMatrix *csr, double *x, double *y, float *elapsed_time) {
    int *IRP = csr->IRP;
    int *JA = csr->JA;
    double *AS = csr->AS;

    int *d_IRP, *d_JA;
    double *d_AS, *d_x, *d_y;
    
    hipMalloc(&d_IRP, (M + 1) * sizeof(int));
    hipMalloc(&d_JA, IRP[M] * sizeof(int));
    hipMalloc(&d_AS, IRP[M] * sizeof(double));
    hipMalloc(&d_x, N * sizeof(double));
    hipMalloc(&d_y, M * sizeof(double));
    
    hipMemcpy(d_IRP, IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_JA, JA, IRP[M] * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_AS, AS, IRP[M] * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);

    int nz = csr->IRP[M];  // numero totale di non zeri
    double avg_nz_row = (double)nz / M; // numero medio di non zeri per riga

    // Configurazione per il calcolo del tempo di esecuzione
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (avg_nz_row < 16) {
        // Lancia il kernel classico thread-per-row
        int blocks = (M + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        hipEventRecord(start, 0);

        spmv_csr_kernel<<<blocks, THREADS_PER_BLOCK>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);

        // Controlla errori di lancio kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
           fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
           exit(EXIT_FAILURE);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(elapsed_time, start, stop);

    } else {
        // Lancia il kernel warp-level: un warp per riga
        int blocks = (M * WARP_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        hipEventRecord(start, 0);

        spmv_csr_warp_kernel<<<blocks, THREADS_PER_BLOCK>>>(M, d_IRP, d_JA, d_AS, d_x, d_y);
        
        // Controlla errori di lancio kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch error (warp-level): %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(elapsed_time, start, stop);
    } 
 
    hipMemcpy(y, d_y, M * sizeof(double), hipMemcpyDeviceToHost);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_IRP);
    hipFree(d_JA);
    hipFree(d_AS);
    hipFree(d_x);
    hipFree(d_y);

}

//HLL

double compute_average_max_nz(const HLLMatrix *hllHost) {
    int numBlocks = hllHost->numBlocks;
    double sum = 0.0;
    for (int b = 0; b < numBlocks; b++) {
        sum += hllHost->blocks[b].maxnz;
    }
    return (numBlocks > 0) ? (sum / numBlocks) : 0.0;
}


/* Kernel CUDA per il prodotto matrice-vettore.
   Ogni thread elabora una riga globale: calcola a quale blocco appartiene e l'indice locale,
   quindi accumula il prodotto per tutti i non-zeri in quella riga. */
__global__ void spmv_hll_kernel(int hackSize, int totalRows, EllpackBlock *d_blocks, const double *d_x, double *d_y) {
    int globalRow = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalRow >= totalRows) return;

    // Determina il blocco e la riga locale in base a hackSize
    int b = globalRow / hackSize;
    int localRow = globalRow % hackSize;
    if (localRow >= d_blocks[b].block_rows)
        return; // nel caso dell'ultimo blocco che contiene meno righe

    double sum = 0.0;
    int maxnz = d_blocks[b].maxnz;
    int rowStart = localRow * maxnz;
    for (int j = 0; j < maxnz; j++) {
        int col = d_blocks[b].JA[rowStart + j];
        if (col != -1) {  // -1 indica una cella vuota
            sum += d_blocks[b].AS[rowStart + j] * d_x[col];
        }
    }
    d_y[globalRow] = sum;
}

// Kernel ottimizzato con warp-level parallelism
__global__ void spmv_hll_kernel_warp(int hackSize, int totalRows, EllpackBlock *d_blocks, const double *d_x, double *d_y) {
    // Ogni warp elabora una riga globale
    int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // indice della riga globale
    int lane   = threadIdx.x % warpSize;  // indice del thread all'interno del warp

    if (warpId >= totalRows) return;

    // Determina il blocco e la riga locale nel blocco HLL
    int b = warpId / hackSize;
    int localRow = warpId % hackSize;
    if (localRow >= d_blocks[b].block_rows)
        return;  // gestione del caso in cui l'ultimo blocco abbia meno righe

    int maxnz = d_blocks[b].maxnz;
    int rowStart = localRow * maxnz;

    // Ogni lane elabora una parte degli elementi della riga:
    double sum = 0.0;
    for (int j = lane; j < maxnz; j += warpSize) {
        int col = d_blocks[b].JA[rowStart + j];
        if (col != -1) {  // -1 indica una cella vuota
            sum += d_blocks[b].AS[rowStart + j] * d_x[col];
        }
    }

    // Riduzione warp-level usando __shfl_down_sync per sommare le parziali
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Il thread lane 0 del warp scrive il risultato della riga
    if (lane == 0) {
        d_y[warpId] = sum;
    }
}


void prod_cuda_hll(const HLLMatrix *hllHost, const double *xHost, double *yHost, int totalRows, float *elapsed_time) {
    int N = hllHost->blocks[0].N;
    double *d_x, *d_y;
    hipMalloc((void**)&d_x, totalRows * sizeof(double));
    hipMalloc((void**)&d_y, totalRows * sizeof(double));
    hipMemcpy(d_x, xHost, N * sizeof(double), hipMemcpyHostToDevice);

    // Allocazione dell'array dei blocchi su device
    EllpackBlock *d_blocks;
    hipMalloc((void**)&d_blocks, hllHost->numBlocks * sizeof(EllpackBlock));

    // Preparo una copia host (temporanea) dei blocchi con i puntatori device
    EllpackBlock *h_blocksDevice = (EllpackBlock *) malloc(hllHost->numBlocks * sizeof(EllpackBlock));
    for (int b = 0; b < hllHost->numBlocks; b++) {
        int sizeBlock = hllHost->blocks[b].block_rows * hllHost->blocks[b].maxnz;
        int *d_JA;
        double *d_AS;
        hipMalloc((void**)&d_JA, sizeBlock * sizeof(int));
        hipMalloc((void**)&d_AS, sizeBlock * sizeof(double));
        // Copia dei dati degli array JA e AS per il blocco corrente
        hipMemcpy(d_JA, hllHost->blocks[b].JA, sizeBlock * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_AS, hllHost->blocks[b].AS, sizeBlock * sizeof(double), hipMemcpyHostToDevice);

        // Imposto il blocco nel vettore temporaneo, con i puntatori aggiornati (device)
        h_blocksDevice[b].block_rows = hllHost->blocks[b].block_rows;
        h_blocksDevice[b].N = hllHost->blocks[b].N;
        h_blocksDevice[b].maxnz = hllHost->blocks[b].maxnz;
        h_blocksDevice[b].JA = d_JA;
        h_blocksDevice[b].AS = d_AS;
    }
    // Copia dell'array dei blocchi (con i puntatori device) su device
    hipMemcpy(d_blocks, h_blocksDevice, hllHost->numBlocks * sizeof(EllpackBlock), hipMemcpyHostToDevice);

    // Costruisco la struttura HLLMatrix sul device
    HLLMatrix hllDevice;
    hllDevice.hackSize = hllHost->hackSize;
    hllDevice.numBlocks = hllHost->numBlocks;
    hllDevice.blocks = d_blocks;
    HLLMatrix *d_hll;
    hipMalloc((void**)&d_hll, sizeof(HLLMatrix));
    hipMemcpy(d_hll, &hllDevice, sizeof(HLLMatrix), hipMemcpyHostToDevice);

    double avg_nz_row = compute_average_max_nz(hllHost);

    // Configurazione per il calcolo del tempo di esecuzione
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    if(avg_nz_row < 16) {
        // Lancio del kernel: un thread per riga globale
        int gridSize = (totalRows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        hipEventRecord(start, 0);

        spmv_hll_kernel<<<gridSize, THREADS_PER_BLOCK>>>(hllHost->hackSize, totalRows, d_blocks, d_x, d_y);

        // Controlla errori di lancio kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
     
        hipEventElapsedTime(elapsed_time, start, stop);

    } else {
        // Lancio del kernel: un warp per riga globale
        int warpsPerBlock = THREADS_PER_BLOCK / WARP_SIZE; // 
        int totalWarps = (totalRows + warpsPerBlock - 1) / warpsPerBlock;

        hipEventRecord(start, 0);

        spmv_hll_kernel_warp<<<totalWarps, THREADS_PER_BLOCK>>>(hllHost->hackSize, totalRows, d_blocks, d_x, d_y);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch error (warp-level): %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
    
        hipEventElapsedTime(elapsed_time, start, stop);

    }

    // Copia del vettore risultato y da device a host
    hipMemcpy(yHost, d_y, totalRows * sizeof(double), hipMemcpyDeviceToHost);

    // Liberazione della memoria device per ciascun blocco
    for (int b = 0; b < hllHost->numBlocks; b++) {
        hipFree(h_blocksDevice[b].JA);
        hipFree(h_blocksDevice[b].AS);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_blocksDevice);
    hipFree(d_blocks);
    hipFree(d_hll);
    hipFree(d_x);
    hipFree(d_y);
}


